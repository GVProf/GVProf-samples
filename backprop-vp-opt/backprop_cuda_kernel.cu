#include "hip/hip_runtime.h"


#ifndef _BACKPROP_CUDA_KERNEL_H_
#define _BACKPROP_CUDA_KERNEL_H_

#include <stdio.h>
#include "backprop.h"
#include "math.h"
#include "hip/hip_runtime.h"


__global__ void
bpnn_layerforward_CUDA(float *input_cuda,
  float *output_hidden_cuda,
  float *input_hidden_cuda,
  float *hidden_partial_sum,
  int in,
  int hid) 
{
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  

   int index_in = HEIGHT * by + ty + 1;
   
   __shared__ float input_node[HEIGHT];
   __shared__ float weight_matrix[HEIGHT][WIDTH];


   if ( tx == 0 )
   input_node[ty] = input_cuda[index_in] ;
   
   __syncthreads();

   weight_matrix[ty][tx] = input_hidden_cuda[index];

   __syncthreads();
   
   weight_matrix[ty][tx] = weight_matrix[ty][tx] * input_node[ty];

   __syncthreads();   
   
   for ( int i = 1 ; i <= __log2f(HEIGHT) ; i++){
 
	   int power_two = __powf(2, i);

	   if( ty % power_two == 0 )
	   weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];

	   __syncthreads();

   }
   
   //__syncthreads();

   input_hidden_cuda[index] = weight_matrix[ty][tx];
   
/*
   for ( unsigned int i = 2 ; i <= HEIGHT ; i *= 2){
 
	   unsigned int power_two = i - 1;

	   if( (ty & power_two) == 0 ) {
		weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];
	   }

   }
   */

   __syncthreads();

   if ( tx == 0 ) {
	   hidden_partial_sum[by * hid + ty] = weight_matrix[tx][ty];
   }

}


__global__ void bpnn_adjust_weights_cuda2(float * delta,   
										 int hid,         
										 float * ly,      
										 int in,          
										 float * w,       
										 float * oldw)  									
{
  
  
   int by = blockIdx.y;

   int tx = threadIdx.x;
   int ty = threadIdx.y;
	
   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  
   int index_y = HEIGHT * by + ty + 1;
   int index_x = tx + 1;
   //eta = 0.3;
   //momentum = 0.3;

   float d = delta[index_x];
   float y = ly[index_y];
   float ww = oldw[index];

   if (ww == 0 && d == 0) {
   } else {
     w[index] += ((ETA * d * y) + (MOMENTUM * ww));
     oldw[index] = ((ETA * d * y) + (MOMENTUM * ww));
   }

   __syncthreads();

   if (ty == 0 && by ==0){
   w[index_x] += ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   oldw[index_x] = ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   }
}

__global__ void bpnn_adjust_weights_cuda(
										 int hid,         
										 float * ly,      
										 int in,          
										 float * w,       
										 float * oldw)  									
{
  
  
   int by = blockIdx.y;

   int tx = threadIdx.x;
   int ty = threadIdx.y;
	
   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;  
   int index_y = HEIGHT * by + ty + 1;
   int index_x = tx + 1;
   //eta = 0.3;
   //momentum = 0.3;

   float ww = oldw[index];

   if (ww == 0) {
   } else {
     w[index] += (MOMENTUM * ww);
     oldw[index] = (MOMENTUM * ww);
   }

   __syncthreads();

   if (ty == 0 && by ==0){
     w[index_x] += MOMENTUM * oldw[index_x];
     oldw[index_x] = MOMENTUM * oldw[index_x];
   }
}

#endif 
