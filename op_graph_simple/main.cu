#include "hip/hip_runtime.h"
#include <cstdio>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../common/common.h"


static const size_t N = 1000;


void init(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    p[i] = i;
  }
}


void output(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    printf("index %zu: %d\n", i, p[i]);
  }
}


__global__
void vecAdd(int *l, int *r, int *p, size_t N) {
  size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < N) {
    p[idx] = l[idx] + r[idx];
  }
}


__global__
void vecAdd_eq(int *l, int *r, int *p, size_t N) {
  size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < N) {
    l[idx] = r[idx];
  }
}


__global__
void vecAdd_odd(int *l, int *r, int *p, size_t N) {
  size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < N && idx % 2) {
    p[idx] = l[idx] + r[idx];
  }
}


int main(int argc, char *argv[]) {
  // Init device
  int device_id = 0;
  if (argc > 1) {
    device_id = atoi(argv[1]);
  }
  cuda_init_device(device_id);

  int l[N], r[N], p[N];
  int *dl, *dr, *dp;

  init(l, N);
  init(r, N);

  RUNTIME_API_CALL(hipMalloc(&dl, N * sizeof(int)));
  RUNTIME_API_CALL(hipMalloc(&dr, N * sizeof(int)));
  RUNTIME_API_CALL(hipMalloc(&dp, N * sizeof(int)));

  RUNTIME_API_CALL(hipMemcpy(dl, l, N * sizeof(int), hipMemcpyHostToDevice));
  RUNTIME_API_CALL(hipMemcpy(dr, r, N * sizeof(int), hipMemcpyHostToDevice));

  // 1. redundant h2d copy
  RUNTIME_API_CALL(hipMemcpy(dl, l, N * sizeof(int), hipMemcpyHostToDevice));

  // 2. redundant d2d copy
  // partial overwrite
  RUNTIME_API_CALL(hipMemcpy(dl, dr, N / 2 * sizeof(int), hipMemcpyDeviceToDevice));

  // non-zero offset redundant write
  RUNTIME_API_CALL(hipMemcpy(dl + N / 2, dl + N / 2, N / 2 * sizeof(int), hipMemcpyDeviceToDevice));

  size_t threads = 256;
  size_t blocks = (N - 1) / threads + 1;

  vecAdd<<<blocks, threads>>>(dl, dr, dp, N);

  // 3. kernel to kernel duplicate
  vecAdd_eq<<<blocks, threads>>>(dp, dp, dp, N);

  // 4. kernel to kernel duplicate, partial write
  vecAdd_odd<<<blocks, threads>>>(dl, dr, dp, N);

  RUNTIME_API_CALL(hipMemcpy(p, dp, N * sizeof(int), hipMemcpyDeviceToHost));

  RUNTIME_API_CALL(hipFree(dl));
  RUNTIME_API_CALL(hipFree(dr));
  RUNTIME_API_CALL(hipFree(dp));

  hipDeviceSynchronize();

  return 0;
}
