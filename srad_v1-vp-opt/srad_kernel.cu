#include "hip/hip_runtime.h"
// BUG IN SRAD APPLICATIONS SEEMS TO BE SOMEWHERE IN THIS CODE, WRONG MEMORY
// ACCESS

// srad kernel
__global__ void srad(fp d_lambda, int d_Nr, int d_Nc, long d_Ne, fp *d_dN, fp *d_dS,
                     fp *d_dE, fp *d_dW, fp d_q0sqr, 
                     fp *d_c,
                      fp *d_I) {

    // indexes
    int bx = blockIdx.x;               // get current horizontal block index
    int tx = threadIdx.x;              // get current horizontal thread index
    int ei = bx * NUMBER_THREADS + tx; // more threads than actual elements !!!
    int row;                           // column, x position
    int col;                           // row, y position

    // variables
    fp d_Jc;
    fp d_dN_loc, d_dS_loc, d_dW_loc, d_dE_loc;
    fp d_c_loc;
    fp d_G2, d_L, d_num, d_den, d_qsqr;

    // figure out row/col location in new matrix
    row = (ei + 1) % d_Nr - 1;     // (0-n) row
    col = (ei + 1) / d_Nr + 1 - 1; // (0-n) column
    if ((ei + 1) % d_Nr == 0) {
        row = d_Nr - 1;
        col = col - 1;
    }

    if (ei < d_Ne) { // make sure that only threads matching jobs run
    int iN, iS, jW, jE;
        if(row == 0) {
        iN = 0;
        }else{
            iN = row -1;
        }
        if(row == d_Nr -1){
            iS = d_Nr -1;
        }else{
            iS = row+1;
        }
        if(col ==0){
            jW = 0;
        }else{
            jW =col -1;
        }
        if(col == d_Nc -1){
            jE = d_Nc - 1;
        }else{
            jE = col +1;
        }
        // directional derivatives, ICOV, diffusion coefficent
        d_Jc = d_I[ei]; // get value of the current element

        // directional derivates (every element of IMAGE)(try to copy to shared
        // memory or temp files)
        d_dN_loc =
            d_I[iN + d_Nr * col] - d_Jc; // north direction derivative
        d_dS_loc =
            d_I[iS + d_Nr * col] - d_Jc; // south direction derivative
        d_dW_loc =
            d_I[row + d_Nr * jW] - d_Jc; // west direction derivative
        d_dE_loc =
            d_I[row + d_Nr * jE] - d_Jc; // east direction derivative

        // normalized discrete gradient mag squared (equ 52,53)
        d_G2 = (d_dN_loc * d_dN_loc + d_dS_loc * d_dS_loc +
                d_dW_loc * d_dW_loc + d_dE_loc * d_dE_loc) /
               (d_Jc * d_Jc); // gradient (based on derivatives)

        // normalized discrete laplacian (equ 54)
        d_L = (d_dN_loc + d_dS_loc + d_dW_loc + d_dE_loc) /
              d_Jc; // laplacian (based on derivatives)

        // ICOV (equ 31/35)
        d_num = (0.5 * d_G2) -
                ((1.0 / 16.0) *
                 (d_L * d_L));    // num (based on gradient and laplacian)
        d_den = 1 + (0.25 * d_L); // den (based on laplacian)
        d_qsqr = d_num / (d_den * d_den); // qsqr (based on num and den)

        // diffusion coefficent (equ 33) (every element of IMAGE)
        d_den = (d_qsqr - d_q0sqr) /
                (d_q0sqr * (1 + d_q0sqr)); // den (based on qsqr and q0sqr)
        d_c_loc = 1.0 / (1.0 + d_den); // diffusion coefficient (based on den)

        // save data to global memory
        d_dN[ei] = d_dN_loc;
        d_dS[ei] = d_dS_loc;
        d_dW[ei] = d_dW_loc;
        d_dE[ei] = d_dE_loc;

        // saturate diffusion coefficent to 0-1 range
        if (d_c_loc < 0.0) {        // if diffusion coefficient < 0
            d_c[ei] = 0.0;
        } else if (d_c_loc < 1.0) {
            d_c[ei] = d_c_loc;
        }
    }
}
